#include "hip/hip_runtime.h"
#include "kernel.h"
#include "defs.h"

// device variables
float2 *d_pos, *d_velo, *d_accel;
float *d_rot, *d_angVelo, *d_angAccel;
float *d_wanderAngle, *d_wanderAngularVelo;
hiprandState_t *d_states;
float *d_configs;

// host variables
float2 *h_pos, *h_velo, *h_accel;
float *h_rot;// , *d_angVelo, *d_angAccel;
float *h_wanderAngle, *h_wanderAngularVelo;
float *h_configs;

const unsigned int threadsPerBlock = 512;
const unsigned int numBlocks = 2;

////////////////////////////////////////////////////////////////////////////////
// CUDA KERNEL FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

__global__ void init_states_kernel(unsigned int seed, hiprandState_t *states) {

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		threadIdx.x, /* the sequence number should be different for each core (unless you want all
					 cores to get the same sequence of numbers for some reason - use thread id! */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[threadIdx.x]);
}

__global__ void copy_pos_kernel(float2 *pos, float2 *newpos, float *rot)
{
	//unsigned int boidIndex = threadIdx.x;
	unsigned int boidIndex = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pointIndex = boidIndex * 6;
	float rot1 = -rot[boidIndex] + 90, rot2 = rot1 - 140, rot3 = rot1 + 140;

	// first triangle
	pos[pointIndex].x = newpos[boidIndex].x;
	pos[pointIndex].y = newpos[boidIndex].y;
	pos[pointIndex + 1].x = newpos[boidIndex].x + cosf(DEG_TO_RAD(rot1)) * BOID_SIZE;
	pos[pointIndex + 1].y = newpos[boidIndex].y + sinf(DEG_TO_RAD(rot1)) * BOID_SIZE;
	pos[pointIndex + 2].x = newpos[boidIndex].x + cosf(DEG_TO_RAD(rot2)) * BOID_SIZE;
	pos[pointIndex + 2].y = newpos[boidIndex].y + sinf(DEG_TO_RAD(rot2)) * BOID_SIZE;

	// second triangle						  					 
	pos[pointIndex + 3].x = newpos[boidIndex].x;
	pos[pointIndex + 3].y = newpos[boidIndex].y;
	pos[pointIndex + 4].x = newpos[boidIndex].x + cosf(DEG_TO_RAD(rot1)) * BOID_SIZE;
	pos[pointIndex + 4].y = newpos[boidIndex].y + sinf(DEG_TO_RAD(rot1)) * BOID_SIZE;
	pos[pointIndex + 5].x = newpos[boidIndex].x + cosf(DEG_TO_RAD(rot3)) * BOID_SIZE;
	pos[pointIndex + 5].y = newpos[boidIndex].y + sinf(DEG_TO_RAD(rot3)) * BOID_SIZE;
}

__global__ void update_kernel(float2 *pos, float2 *velo, float2  *accel, float *rot,
	float *wanderAngle, float *wanderAngularVelo, hiprandState_t *states, float *configs)
{
	//unsigned int index = threadIdx.x;
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

	//wanderBehavior2(index, pos, accel, velo, rot, wanderAngle, wanderAngularVelo, states);
	flockingBehavior(index, pos, velo, accel, configs);

	///////////////physics
	applyAcceleration(index, velo, accel);
	lookWhereYourGoing(index, pos, velo, rot);
	applyVelocity(index, pos, velo);

	// hiprand test http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
	//hiprand_init(673,0,0,&state);
	/*pos[index].x = hiprand(&states[index]) % window_width;
	pos[index].y = hiprand(&states[index]) % window_height;*/
}

////////////////////////////////////////////////////////////////////////////////
// FLOCKING BEHAVIOUR FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

__device__ void flockingBehavior(unsigned int index, float2 *pos, float2 *velo, float2 *accel, float *configs) {
	// store the positions in a shared buffer
	__shared__ float2 posBuffer[1024];
	__shared__ float2 veloBuffer[1024];
	posBuffer[index].x = pos[index].x;
	posBuffer[index].y = pos[index].y;
	veloBuffer[index].x = velo[index].x;
	veloBuffer[index].y = velo[index].y;

	__syncthreads(); // all the threads must be synced here, so the buffers are filled!

	// implement alignment, cohesion and seperation vectors
	float2 alignment = make_float2(0.f, 0.f);
	float2 cohesion = make_float2(0.f, 0.f);
	float2 seperation = make_float2(0.f, 0.f);

	int numNeighbors = 0;
	for (int i = 0; i < 1024; ++i) {
		// skip yourself
		if (i == index)
			continue;

		// calculate squareDistance
		float dx = posBuffer[index].x - posBuffer[i].x;
		float dy = posBuffer[index].y - posBuffer[i].y;
		float sqrDistance = dx * dx + dy * dy;

		// for every close neighbor
		if (sqrDistance < SQR_LOOK_DISTANCE) {
			numNeighbors++;

			alignment.x += veloBuffer[i].x;
			alignment.y += veloBuffer[i].y;
			cohesion.x += posBuffer[i].x;
			cohesion.y += posBuffer[i].y;
			seperation.x += dx;
			seperation.y += dy;
		}
	}//endfor

	 // no neighbors found?
	if (numNeighbors == 0) {
		alignment.x = 0.f;
		alignment.y = 0.f;
		cohesion.x = 0.f;
		cohesion.y = 0.f;
		seperation.x = 0.f;
		seperation.y = 0.f;
	}
	else {
		/*alignment.x /= numNeighbors;
		alignment.y /= numNeighbors;*/
		cohesion.x /= numNeighbors;
		cohesion.y /= numNeighbors;
		cohesion = make_float2(cohesion.x - posBuffer[index].x, cohesion.y - posBuffer[index].y);
		/*seperation.x /= numNeighbors;
		seperation.y /= numNeighbors;*/

		alignment = normalize2(alignment);
		cohesion = normalize2(cohesion);
		seperation = normalize2(seperation);
	}

	float2 desiredVelo;
	desiredVelo.x = configs[0] * alignment.x + configs[1] * cohesion.x + configs[2] * seperation.x;
	desiredVelo.y = configs[0] * alignment.y + configs[1] * cohesion.y + configs[2] * seperation.y;
	desiredVelo = normalize2(desiredVelo);
	desiredVelo.x *= MAX_VELOCITY;
	desiredVelo.y *= MAX_VELOCITY;
	accel[index].x += (desiredVelo.x - veloBuffer[index].x);
	accel[index].y += (desiredVelo.y - veloBuffer[index].y);
}

// doesnt work, always aligns on diagonal line
__device__ void wanderBehavior(unsigned int index, float2 *pos, float2 *accel, float2 *velo, float *rot, float *wanderAngle, float *wanderAngularVelo, hiprandState_t *states) {
	// wander behaviour
	float2 circleCenter = make_float2(0.0f, 0.0f),
		circlePoint = make_float2(0.0f, 0.0f),
		desiredPos = make_float2(0.0f, 0.0f),
		desiredVelo = make_float2(0.0f, 0.0f);
	// calculate circle center
	float currentRotation = DEG_TO_RAD(rot[index]);
	circleCenter.x = pos[index].x + CENTER_OFFSET * cosf(currentRotation);
	circleCenter.y = pos[index].y + CENTER_OFFSET * sinf(currentRotation);
	float circleAngle = wanderAngle[index] + DEG_TO_RAD(rot[index]);
	// calculate desired position
	circlePoint.x = WANDER_RADIUS * cos(circleAngle);
	circlePoint.y = WANDER_RADIUS * sin(circleAngle);
	desiredPos.x = circleCenter.x + circlePoint.x;
	desiredPos.y = circleCenter.y + circlePoint.y;
	// calculate desired velo and resulting acceleration
	desiredVelo.x = desiredPos.x - pos[index].x;
	desiredVelo.y = desiredPos.y - pos[index].y;
	accel[index].x = desiredVelo.x - velo[index].x;
	accel[index].y = desiredVelo.y - velo[index].y;

	// move the circle point randomly on the circular path
	// calculate a randomized acceleration for the circle point
	float wanderAngularAccel = (0.2*double(hiprand(&states[index])) / double(RAND_MAX) - 0.1);
	wanderAngularVelo[index] += 0.5f * wanderAngularAccel;
	CLAMP(-MAX_WANDER_VELO, wanderAngularVelo[index], MAX_WANDER_VELO);
	wanderAngle[index] += 0.5f * wanderAngularVelo[index];
}

// still problem with random numbers
__device__ void wanderBehavior2(unsigned int index, float2 *pos, float2 *accel, float2 *velo, float *rot, float *wanderAngle, float *wanderAngularVelo, hiprandState_t *states) {
	// wander behaviour from here: https://gamedevelopment.tutsplus.com/tutorials/understanding-steering-behaviors-wander--gamedev-1624
	float2 circleCenter = make_float2(0.0f, 0.0f),
		displacement = make_float2(0.0f, -1.f);
	// calculate circle center
	circleCenter.x = velo[index].x;
	circleCenter.y = velo[index].y;
	circleCenter = normalize2(circleCenter);
	circleCenter.x *= CENTER_OFFSET;
	circleCenter.y *= CENTER_OFFSET;

	// calculate displacement force
	displacement.x = cosf(wanderAngle[index]) * WANDER_RADIUS;
	displacement.y = sinf(wanderAngle[index]) * WANDER_RADIUS;

	accel[index].x = circleCenter.x + displacement.x;
	accel[index].y = circleCenter.y + displacement.y;

	// move the circle point randomly on the circular path by changing the wanderAngle
	float wanderAngularAccel = (0.2*double(hiprand(&states[index])) / double(RAND_MAX) - 0.1);
	wanderAngularVelo[index] += 0.5f * wanderAngularAccel;
	CLAMP(-MAX_WANDER_VELO, wanderAngularVelo[index], MAX_WANDER_VELO);
	wanderAngle[index] += 0.5f * wanderAngularVelo[index];
}

////////////////////////////////////////////////////////////////////////////////
// PHYSICS FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

__device__ void applyVelocity(unsigned int index, float2 *pos, float2 *velo) {
	// apply velocity
	pos[index].x += DELTA_TIME * velo[index].x;
	pos[index].y += DELTA_TIME * velo[index].y;

	// cap the position
	if (pos[index].x > WINDOW_WIDTH) {
		pos[index].x -= WINDOW_WIDTH;
	}
	if (pos[index].x < 0) {
		pos[index].x += WINDOW_WIDTH;
	}
	if (pos[index].y > WINDOW_HEIGHT) {
		pos[index].y -= WINDOW_HEIGHT;
	}
	if (pos[index].y < 0) {
		pos[index].y += WINDOW_HEIGHT;
	}
}
__device__ void lookWhereYourGoing(unsigned int index, float2 *pos, float2 *velo, float *rot) {
	if (length2(velo[index]) > EPSILON) {
		rot[index] = RAD_TO_DEG(atan2(velo[index].x, velo[index].y));
	}
}
__device__ void applyAcceleration(unsigned int index, float2 *velo, float2 *accel) {
	// cap acceleration
	if (length2(accel[index]) > MAX_ACCELERATION) {
		velo[index] = normalize2(velo[index]);
		velo[index].x *= MAX_ACCELERATION;
		velo[index].y *= MAX_ACCELERATION;
	}

	// apply acceleration
	velo[index].x += DELTA_TIME * accel[index].x;
	velo[index].y += DELTA_TIME * accel[index].y;

	// cap velocity
	if (length2(velo[index]) > MAX_VELOCITY) {
		velo[index] = normalize2(velo[index]);
		velo[index].x *= MAX_VELOCITY;
		velo[index].y *= MAX_VELOCITY;
	}
}

////////////////////////////////////////////////////////////////////////////////
// HELPER FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

__device__ float2 normalize2(float2 p)
{
	float length = sqrt(p.x * p.x + p.y * p.y);
	float px = p.x;
	float py = p.y;
	if (length <= FLT_EPSILON) {
		px = 0.f;
		py = 0.f;
	}
	else {
		px /= length;
		py /= length;
	}
	return make_float2(px, py);
}
__device__ float length2(float2 p) {
	return sqrt(p.x * p.x + p.y * p.y);
}
__device__ float sqrLength2(float2 p) {
	return p.x * p.x + p.y * p.y;
}

////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

// called once, allocates all the memory on the cuda device
void init_kernel() {
	// allocate host arrays
	hipHostAlloc(&h_pos, sizeof(float2) * NUMBER_OF_BOIDS, hipHostMallocDefault);
	hipHostAlloc(&h_velo, sizeof(float2) * NUMBER_OF_BOIDS, hipHostMallocDefault);
	hipHostAlloc(&h_accel, sizeof(float2) * NUMBER_OF_BOIDS, hipHostMallocDefault);
	hipHostAlloc(&h_rot, sizeof(float) * NUMBER_OF_BOIDS, hipHostMallocDefault);
	hipHostAlloc(&h_wanderAngle, sizeof(float) * NUMBER_OF_BOIDS, hipHostMallocDefault);
	hipHostAlloc(&h_wanderAngularVelo, sizeof(float) * NUMBER_OF_BOIDS, hipHostMallocDefault);

	// init host array
	for (int i = 0; i < NUMBER_OF_BOIDS; ++i) {
		h_pos[i].x = rand() % WINDOW_WIDTH;
		h_pos[i].y = rand() % WINDOW_HEIGHT;
		h_velo[i].x = 0.f;
		h_velo[i].y = 0.f;
		h_accel[i].x = (2.0*float(rand()) / float(RAND_MAX) - 1.0f) * MAX_ACCELERATION;
		h_accel[i].y = (2.0*float(rand()) / float(RAND_MAX) - 1.0f) * MAX_ACCELERATION;
		h_rot[i] = rand() % 360;
		h_wanderAngle[i] = (rand() % 100) / 100.f * 2 * M_PI;
		h_wanderAngularVelo[i] = 0.1*(2.0f*double(rand() + i) / double(RAND_MAX) - 1.0f);
	}

	// allocate device arrays
	checkCudaErrors(hipMalloc(&d_pos, sizeof(float2) * NUMBER_OF_BOIDS));
	checkCudaErrors(hipMalloc(&d_velo, sizeof(float2) * NUMBER_OF_BOIDS));
	checkCudaErrors(hipMalloc(&d_accel, sizeof(float2) * NUMBER_OF_BOIDS));
	checkCudaErrors(hipMalloc(&d_rot, sizeof(float) * NUMBER_OF_BOIDS));
	checkCudaErrors(hipMalloc(&d_wanderAngle, sizeof(float) * NUMBER_OF_BOIDS));
	checkCudaErrors(hipMalloc(&d_wanderAngularVelo, sizeof(float) * NUMBER_OF_BOIDS));

	copy_host_to_device();

	// allocate space for random states
	checkCudaErrors(hipMalloc(&d_states, sizeof(hiprandState_t) * NUMBER_OF_BOIDS));
	init_states_kernel << <numBlocks, threadsPerBlock >> >(time(0), d_states);

	// allocate and init configuration stuff
	hipHostAlloc(&h_configs, sizeof(float) * NUM_OF_CONFIG_VARS, hipHostMallocDefault);
	for (int i = 0; i < NUM_OF_CONFIG_VARS; ++i) {
		h_configs[i] = 1.f;
	}
	checkCudaErrors(hipMalloc(&d_configs, sizeof(float) * NUM_OF_CONFIG_VARS));
	checkCudaErrors(hipMemcpy(d_configs, h_configs, sizeof(float) * NUM_OF_CONFIG_VARS, hipMemcpyHostToDevice));
}

// used to reset the positions
void copy_host_to_device() {
	// copy to device
	checkCudaErrors(hipMemcpy(d_pos, h_pos, sizeof(float2) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_velo, h_velo, sizeof(float2) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_accel, h_accel, sizeof(float2) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_rot, h_rot, sizeof(float) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wanderAngle, h_wanderAngle, sizeof(float) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_wanderAngularVelo, h_wanderAngularVelo, sizeof(float) * NUMBER_OF_BOIDS, hipMemcpyHostToDevice));
}

void update_configs(float *configs) {
	for (int i = 0; i < NUM_OF_CONFIG_VARS; ++i) {
		h_configs[i] = configs[i];
	}
	checkCudaErrors(hipMemcpy(d_configs, h_configs, sizeof(float) * NUM_OF_CONFIG_VARS, hipMemcpyHostToDevice));
}

// launches the kernel that is doing the simulation step
void launch_update_kernel() {
	update_kernel << <numBlocks, threadsPerBlock >> >(d_pos, d_velo, d_accel, d_rot, d_wanderAngle,
		d_wanderAngularVelo, d_states, d_configs);
}

// gets called to update the vbo
void launch_vbo_kernel(float2 *pos)
{
	//simple_vbo_kernel<<<1,1024>>>(pos, goal, weights);
	copy_pos_kernel << <numBlocks, threadsPerBlock >> >(pos, d_pos, d_rot);
}

// cleans up all the allocated memory on the device
void cleanupKernel() {
	hipHostFree(h_pos);
	hipHostFree(h_velo);
	hipHostFree(h_accel);
	hipHostFree(h_rot);
	hipHostFree(h_wanderAngle);
	hipHostFree(h_wanderAngularVelo);
	hipHostFree(h_configs);

	hipFree(d_pos);
	hipFree(d_velo);
	hipFree(d_accel);
	hipFree(d_rot);
	hipFree(d_wanderAngle);
	hipFree(d_wanderAngularVelo);
	hipFree(d_states);
	hipFree(d_configs);
}